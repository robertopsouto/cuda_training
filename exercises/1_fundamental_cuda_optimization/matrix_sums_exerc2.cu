
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <bits/stdc++.h>

// Error checking macro
#define cudaCheckErrors(msg) \
    do { \
        hipError_t __err = hipGetLastError(); \
        if (__err != hipSuccess) { \
            fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
                msg, hipGetErrorString(__err), \
                __FILE__, __LINE__); \
            fprintf(stderr, "*** FAILED - ABORTING\n"); \
            exit(1); \
        } \
    } while (0)

// Matrix side dimension
const size_t DSIZER = 16384;
const size_t DSIZEC = 32768;
// CUDA maximum is 1024
const int block_size = 256;  

// Matrix row-sum kernel
__global__ void row_sums(const float *A, float *sums, size_t ds)
{
  // Create typical 1D thread index from built-in variables
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < ds)
  {
    float sum = 0.0f;
    // Write a for loop that will cause the thread
    // to iterate across a row, keeeping a running sum,
    // and write the result to sums
    for (size_t i = 0; i < ds; i++)
      //sum += A[i*blockDim.x + threadIdx.x];
      sum += A[blockIdx.x*blockDim.x + i];
    sums[idx] = sum;
  }
}

// Matrix column-sum kernel
__global__ void column_sums(const float *A, float *sums, size_t ds){
  // create typical 1D thread index from built-in variables
  int idx = blockIdx.x*blockDim.x + threadIdx.x;
  if (idx < ds)
  {
    float sum = 0.0f;
    // Write a for loop that will cause the thread 
    // to iterate down a column, keeeping a running sum, 
    // and write the result to sums
    for (size_t i = 0; i < ds; i++)
      //sum += A[blockIdx.x*blockDim.x + i];
      sum += A[i*blockDim.x + threadIdx.x];
    sums[idx] = sum;
    }
}

bool validate(float *data, size_t sz)
{
  for (size_t i = 0; i < sz; i++)
    if (data[i] != (float)sz)
    {
      printf("results mismatch at %lu, was: %f, should be: %f\n", i, data[i], (float)sz); return false;
    }
    return true;
}

int main()
{
  float *h_A, *h_sumsR, *h_sumsC, *d_A, *d_sumsR, *d_sumsC;
  // Allocate space for data in host memory
  h_A = new float[DSIZER*DSIZEC];
  h_sumsR = new float[DSIZER]();
  h_sumsC = new float[DSIZEC]();
  
  // Initialize matrix in host memory
  for (int i = 0; i < DSIZER*DSIZEC; i++)
    h_A[i] = 1.0f;
    //h_A[i] = rand();

  
  // Allocate device space for A
  hipMalloc(&d_A, DSIZER*DSIZEC*sizeof(float));
  
  // Allocate device space for vector d_sums FIXME
  hipMalloc(&d_sumsR, DSIZER*sizeof(float));
  hipMalloc(&d_sumsC, DSIZEC*sizeof(float));
  
  cudaCheckErrors("hipMalloc failure"); // error checking
  
  // Copy matrix A to device
  hipMemcpy(d_A, h_A, DSIZER*DSIZEC*sizeof(float), hipMemcpyHostToDevice);
  cudaCheckErrors("hipMemcpy H2D failure");
  
  // CUDA processing sequence step 1 is complete
  row_sums<<<(DSIZER+block_size-1)/block_size, block_size>>>(d_A, d_sumsR, DSIZER);
  cudaCheckErrors("kernel launch failure");
  
  // CUDA processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sumsR, d_sumsR, DSIZER*sizeof(float), hipMemcpyDeviceToHost);
  
  // CUDA processing sequence step 3 is complete
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  if (!validate(h_sumsR, DSIZER)) return -1; 
  
  printf("row sums correct!\n");
  
  hipMemset(d_sumsC, 0, DSIZEC*sizeof(float));
  
  column_sums<<<(DSIZEC+block_size-1)/block_size, block_size>>>(d_A, d_sumsC, DSIZEC);
  cudaCheckErrors("kernel launch failure");
  
  // CUDA processing sequence step 2 is complete
  // copy vector sums from device to host:
  hipMemcpy(h_sumsC, d_sumsC, DSIZEC*sizeof(float), hipMemcpyDeviceToHost);
  
  //CUDA processing sequence step 3 is complete
  cudaCheckErrors("kernel execution failure or hipMemcpy H2D failure");
  
  if (!validate(h_sumsC, DSIZEC)) return -1; 
  printf("column sums correct!\n");
  
  return 0;
}
